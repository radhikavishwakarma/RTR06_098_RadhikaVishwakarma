#include <stdio.h>

int main(void)
{
    // function declarations
    void PrintCUDADeviceProperties(void);

    // code
    PrintCUDADeviceProperties();
}

void PrintCUDADeviceProperties(void)
{
    // code
    printf("CUDA INFORMATION :\n");
    printf
        ("=================================================================\n");

    hipError_t ret_cuda_rt;
    int dev_count;
    ret_cuda_rt = hipGetDeviceCount(&dev_count);
    if (ret_cuda_rt != hipSuccess)
    {
        printf("CUDA Runtime API Error - hipGetDeviceCount() Failed Due To %s\n", hipGetErrorString(ret_cuda_rt));
    }
    else if (dev_count == 0)
    {
        printf("There Is No CUDA Supported Device On This System.\n");
        return;
    }
    else
    {
        printf("Total Number Of CUDA Supporting GPU Device/Devices On This System : %d\n", dev_count);
        for (int i = 0; i < dev_count; i++)
        {
            hipDeviceProp_t dev_prop;
            int driverVersion = 0, runTimeVersion = 0;

            ret_cuda_rt = hipGetDeviceProperties(&dev_prop, i);
            if (ret_cuda_rt != hipSuccess)
            {
                printf("%s in %s at line %d\n", hipGetErrorString (ret_cuda_rt), __FILE__, __LINE__);
                return;
            }

            printf("\n");
            hipDriverGetVersion(&driverVersion);
            hipRuntimeGetVersion(&runTimeVersion);
            printf("*******CUDA DRIVER AND RUNTIME INFORMATION*******\n");
            printf("=========================================================\n");
            printf("CUDA Driver Version   : %d.%d\n", driverVersion / 1000, (driverVersion % 100) / 10);
            printf("CUDA Runtime Version   : %d.%d\n", runTimeVersion / 1000, (runTimeVersion % 100) / 10);
            printf("\n");
            printf("=========================================================\n");
            printf("******* GPU DEVICE GENERAL INFORMATION *******\n");
            printf("=========================================================\n");
            printf("GPU Device Number      : %d\n", i);
            printf("GPU Device Name        : %s\n", dev_prop.name);
            printf("GPU Device Compute Capability   : %d.%d\n", dev_prop.major, dev_prop.minor);
            printf("GPU Device ClockRate      : %d\n", dev_prop.clockRate);
            printf("GPU Device Type                          :");
            if (dev_prop.integrated)
            
                printf("Intergrated ( On-Board )\n");
            else
                printf("Discrete ( Card )\n");
            printf("\n");
            printf("******* GPU DEVICE GENERAL INFORMATION *******\n");
            printf("=========================================================\n");
            printf("GPU Device Total Memory        : %.0f GB = %.0f MB = %llu Bytes\n", ((float)dev_prop.totalGlobalMem / 1048576.0f) / 1024.0f, (float)dev_prop.totalGlobalMem / 1048576.0f, (unsigned long long) dev_prop.totalGlobalMem);
            printf("GPU Device Shared Memory Per SMProcessor     : %lu\n", (unsigned long)dev_prop.sharedMemPerBlock);
            printf("\n");
            printf("******** GPU DEVICE MULTIPROCESSOR INFORMATION ********\n");
            printf("=========================================================\n");
            printf("GPU Device Number Of SMProcessors         : %d\n", dev_prop.multiProcessorCount);
            printf("GPU Device Number Of Registers Per SMProcessor         :%d\n", dev_prop.regsPerBlock);
            printf("\n");
            printf("******* GPU DEVICE THREAD INFORMATION *******\n");
            printf("=========================================================\n");
            printf("GPU Device Maximum Number Of Threads Per SMProcessor  : %d\n", dev_prop.maxThreadsPerMultiProcessor);
            printf("GPU Device Maximum Number Of Threads Per Block : %d\n", dev_prop.maxThreadsPerBlock);
            printf("GPU Device Threads In Warp         : %d\n", dev_prop.warpSize);
            printf("GPU Device Maximum Thread Dimensions          : ( %d, %d, %d )\n", dev_prop.maxThreadsDim[0], dev_prop.maxThreadsDim[1], dev_prop.maxThreadsDim[2]);
            printf("GPU Device Maximum Grid Dimensions          : ( %d, %d, %d )\n", dev_prop.maxGridSize[0], dev_prop.maxGridSize[1], dev_prop.maxGridSize[2]);
            printf("\n");
            printf("*********** GPU DEVICE DRIVER INFORMATION ************\n");
            printf("=========================================================\n");
            printf("GPU Device has ECC support         : %s \n", dev_prop.ECCEnabled ? "Enabled" : "Disabled");

            #if defined(WIN32) || defined(WIN32) || defined(WIN64) || defined(_WIN64)
                        printf("GPU Device CUDA Driver Mode ( TCC Or WDDM)       : %s\n", dev_prop.tccDriver ? "TCC ( Tesla Compute Cluster Driver )" : "WDDM ( Windows Display Driver Model)");
            #endif
                        printf
                          ("**************************************************************************************************************************************************\n"); 
    
            
        }
        
    }
      
}