#include "hip/hip_runtime.h"
// header fils
#include <stdio.h>

// cuda headers
#include <hip/hip_runtime.h>
#include "helper_timer.h"

// macros
#define BLOCK_WIDTH 32

// global variables
int* hostA = NULL;
int* hostB = NULL;
int* hostC = NULL;
int* gold = NULL;

int* deviceA = NULL;
int* deviceB = NULL;
int* deviceC = NULL;

float timeOnCPU = 0.0f;
float timeOnGPU = 0.0f;

// CUDA Kernel
__global__ void matMulGPU(int *A, int *B, int *C, int numARows, int numAColumns, int numBColumns, int numCColumns)
{
    // variable declaration
    int row = blockIdx.y* blockDim.y + threadIdx.y;
    int column = blockIdx.x* blockDim.x + threadIdx.x;
    
    // code
    if((row < numARows) && (column < numBColumns))
    {
        int value = 0.0f;
        for (int k = 0; k < numAColumns; k++)
        {
            int a = A[row * numAColumns + k];
            int b = B[k * numBColumns + column];
            value += a*b;
        }
        C[row * numCColumns + column] = value;
    }
}

// entry -point function
int main(void)
{
    // function declarations
    void InitA(int *data, int , int);
    void InitB(int *data, int , int);
    void matMulCPU(int*, int*, int*, int, int, int, int);
    void cleanup(void);
    
    // variable declaration
    int numARows = BLOCK_WIDTH;
    int numAColumns = BLOCK_WIDTH;
    int numBRows = BLOCK_WIDTH;
    int numBColumns = BLOCK_WIDTH;
    int numCRows = BLOCK_WIDTH;
    int numCColumns = BLOCK_WIDTH;

    int numGoldRows = BLOCK_WIDTH;
    int numGoldColumns = BLOCK_WIDTH;

    int sizeA = numARows * numAColumns * sizeof(int);
    int sizeB = numBRows * numBColumns * sizeof(int);
    int sizeC = numCRows * numCColumns * sizeof(int);
    int sizeGold = numGoldRows * numGoldColumns * sizeof(int);

    hipError_t result = hipSuccess;

    // code
    //host memory allcation
    hostA = (int *)malloc(sizeA);
    if(hostA == NULL)
    {
        printf("Host Memory allocation is failed for hostA array\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    hostB = (int *)malloc(sizeB);
    if(hostB == NULL)
    {
        printf("Host Memory allocation is failed for hostB array\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    hostC = (int *)malloc(sizeC);
    if(hostC == NULL)
    {
        printf("Host Memory allocation is failed for hostC array\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    gold = (int *)malloc(sizeGold);
    if(gold == NULL)
    {
        printf("Host Memory allocation is failed for gold array\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    // printing matrix dimensions and sizes
    printf("The dimensions of Matrix 'hostA' are : %d x %d\n", numARows, numAColumns);
    printf("The dimensions of Matrix 'hostA' are : %d x %d\n", numBRows, numBColumns);
    printf("The dimensions of Matrix 'hostA' are : %d x %d\n", numCRows, numCColumns);
    printf("The dimensions of Matrix 'hostA' are : %d x %d\n", numGoldRows, numGoldColumns);

    printf("Size of Matrix hostA = %d\n", sizeA);
    printf("Size of Matrix hostB = %d\n", sizeB);
    printf("Size of Matrix hostC = %d\n", sizeC);
    printf("Size of Matrix hostGold = %d\n", sizeGold);

    // fill source matrices
    InitA(hostA, numARows, numAColumns);
    InitB(hostB, numBRows, numBColumns);

    // device memory allcation
    result = hipMalloc((void**)&deviceA, sizeA);
    if(result != hipSuccess)
    {
        printf("Device Memory Allocation is failed for deviceA array\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    result = hipMalloc((void**)&deviceB, sizeB);
    if(result != hipSuccess)
    {
        printf("Device Memeory Allocation is failed for deviceB array\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    result = hipMalloc((void**)&deviceC, sizeC);
    if(result != hipSuccess)
    {
        printf("Device Memeory Allocation is failed for deviceC array\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    //copy data from host array into device arrays
    result = hipMemcpy(deviceA,hostA,sizeA,hipMemcpyHostToDevice);
    if(result != hipSuccess)
    {
        printf("Host to device Data Copy is failed for deviceA array\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    result = hipMemcpy(deviceB,hostB,sizeA,hipMemcpyHostToDevice);
    if(result != hipSuccess)
    {
        printf("Host to device Data Copy is failed for deviceB array\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    // CUDA Kernel Configuration
    dim3 dimGrid = dim3(ceil((int)numBColumns / (int)BLOCK_WIDTH), ceil((int)numARows/(int)BLOCK_WIDTH), 1);
    dim3 dimBlock = dim3(BLOCK_WIDTH, 1, 1);

    // CUDA Kernel for Matrix Multiplication
    StopWatchInterface* timer = NULL;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);
    
    matMulGPU <<<dimGrid, dimBlock >>>(deviceA, deviceB, deviceC, numARows, numAColumns, numBRows, numBColumns);
    sdkStopTimer(&timer);

    timeOnGPU = sdkGetTimerValue(&timer);
    sdkDeleteTimer(&timer);
    timer = NULL;

    // Copy data from device array into host array
    result = hipMemcpy(hostC, deviceC, sizeC, hipMemcpyDeviceToHost);
    if(result != hipSuccess)
    {
        printf("Device to Host Data Copy is failed for hostC array..\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    // matrix multiplication on host
    matMulCPU(hostA, hostB, gold, numARows, numAColumns, numBColumns, numCColumns);

    // comparison
    int breakValue = -1;
    bool bAccuracy = true;

    for(int i = 0; i < numCRows * numCColumns; i++)
    {
        int val1 = gold[i];
        int val2 = hostC[i];
        if (val1 != val2)
        {
            bAccuracy = false;
            breakValue = i;
            break;
        }        
    }

    char str[128];
    if(bAccuracy == false)
    {
        sprintf(str, "Comparison of CPU and GPU Matrix Multiplication is not accurate at array index %d\n", breakValue);
    }
    else
    {
        sprintf(str, "Comparison of CPU and GPU Matrix Multiplication is accurate at array index %d\n", breakValue);
    }

    printf("Time taken for Matrix Multiplication in CPU = %.6f\n", timeOnCPU);
    printf("Time taken for Matrix Multiplication in GPU = %.6f\n", timeOnGPU);
    printf("%s\n", str);

    //cleanup
    cleanup();

    return 0;
}

void InitA(int *data, int row, int col)
{
    int num=1;
    // code
    for (int i = 0; i < row; i++)
    {
        for (int j = 0; j < col; j++)
        {
            *(data + 1 * col + j) = num;
            num++;
        }
    }
}

void InitB(int *data, int row, int col)
{
    int num=BLOCK_WIDTH;
    // code
    for (int i = 0; i < row; i++)
    {
        for (int j = 0; j < col; j++)
        {
            *(data + 1 * col + j) = num;
            num++;
        }
    }
}

void matMulCPU(int* A, int* B, int* C, int numARows, int numAColumns, int numBColumns, int numCColumns)
{
    // code
    StopWatchInterface* timer = NULL;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

    for (int i = 0; i < numARows; ++i)
    {
        for (int j = 0; j < numBColumns; ++j)
        {
            int value = 0;
            for (int k = 0; k < numAColumns; ++k)
            {
                int a = A[i * numAColumns + k];
                int b = B[k * numBColumns + j];
                value += a * b;
            }
            C[i * numCColumns + j] = value;
        }
    }

    sdkStopTimer(&timer);
    timeOnCPU = sdkGetTimerValue(&timer);
    sdkDeleteTimer(&timer);
    timer = NULL;
}

void cleanup(void)
{
    //code
    if(deviceC)
    {
        hipFree(deviceC);
        deviceC = NULL;
    }

    if(deviceB)
    {
        hipFree(deviceB);
        deviceB = NULL;
    }

    if(deviceA)
    {
        hipFree(deviceA);
        deviceA = NULL;
    }

    if(gold)
    {
        free(gold);
        gold = NULL;
    }

    if(hostC)
    {
        free(hostC);
        hostC = NULL;
    }

    if(hostB)
    {
        free(hostB);
        hostB= NULL;
    }

    if(hostA)
    {
        free(hostA);
        hostA= NULL;
    }
}
